#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        printf("0\n");
        return 0;
    }

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    printf("%d%d\n", prop.major, prop.minor);
    return 0;
}