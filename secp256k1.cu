/******************************************************************************************************
 * This file is part of the Pollard's Rho distribution: (https://github.com/lucaselblanc/pollardsrho) *
 * Copyright (c) 2024, 2025 Lucas Leblanc.                                                            *
 * Distributed under the MIT software license, see the accompanying.                                  *
 * file COPYING or https://www.opensource.org/licenses/mit-license.php.                               *
 ******************************************************************************************************/

/*****************************************
 * Pollard's Rho Algorithm for SECP256K1 *
 * Written by Lucas Leblanc              *
******************************************/

/* --- AINDA EM TESTES --- */

#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>


__constant__ uint64_t P_CONST[4] = {
    0xFFFFFFFEFFFFFC2FULL,
    0xFFFFFFFFFFFFFFFFULL,
    0xFFFFFFFFFFFFFFFFULL,
    0xFFFFFFFFFFFFFFFFULL
};

__constant__ uint64_t N_CONST[4] = {
    0xBFD25E8CD0364141ULL,
    0xBAAEDCE6AF48A03BULL,
    0xFFFFFFFFFFFFFFFEULL,
    0xFFFFFFFFFFFFFFFFULL
};

__constant__ uint64_t GX_CONST[4] = {
    0x59F2815B16F81798ULL,
    0x029BFCDB2DCE28D9ULL,
    0x55A06295CE870B07ULL,
    0x79BE667EF9DCBBACULL
};

__constant__ uint64_t GY_CONST[4] = {
    0x9C47D08FFB10D4B8ULL,
    0xFD17B448A6855419ULL,
    0x5DA4FBFC0E1108A8ULL,
    0x483ADA7726A3C465ULL
};

__constant__ uint64_t R_MOD_P[4] = {
    0x00000001000003D1ULL,
    0x0000000000000000ULL,
    0x0000000000000000ULL,
    0x0000000000000000ULL
};

__constant__ uint64_t R2_MOD_P[4] = {
    0x000007A2000E90A1ULL,
    0x0000000100000000ULL,
    0x0000000000000000ULL,
    0x0000000000000000ULL
};

__constant__ uint64_t R2_MOD_N[4] = {
    0x896CF21467D7D140ULL,
    0x741496C20E7CF878ULL,
    0xE697F5E45BCD07C6ULL,
    0x9D671CD581C69BC5ULL
};

__constant__ uint64_t MU_P = 0xD2253531ULL;
__constant__ uint64_t MU_N = 0x5588B13FULL;

__constant__ uint64_t ZERO[4]  = {0ULL, 0ULL, 0ULL, 0ULL};
__constant__ uint64_t ONE[4]   = {1ULL, 0ULL, 0ULL, 0ULL};
__constant__ uint64_t TWO[4]   = {2ULL, 0ULL, 0ULL, 0ULL};
__constant__ uint64_t THREE[4] = {3ULL, 0ULL, 0ULL, 0ULL};
__constant__ uint64_t SEVEN[4] = {7ULL, 0ULL, 0ULL, 0ULL};

__constant__ uint64_t ONE_MONT[4] = {0x00000001000003D1ULL, 0x0000000000000000ULL, 0x0000000000000000ULL, 0x0000000000000000ULL};

__constant__ uint64_t SEVEN_MONT[4] = {0x0000000700001A97ULL, 0x0000000000000000ULL, 0x0000000000000000ULL, 0x0000000000000000ULL};

typedef struct {
    uint64_t x[4];
    uint64_t y[4];
    int infinity;
} ECPoint;

typedef struct {
    uint64_t X[4];
    uint64_t Y[4];
    uint64_t Z[4];
    int infinity;
} ECPointJacobian;

__device__ int bignum_cmp(const uint64_t *a, const uint64_t *b) {
    for (int i = 3; i >= 0; i--) {
        if (a[i] > b[i]) return 1;
        if (a[i] < b[i]) return -1;
    }
    return 0;
}

__device__ int bignum_is_zero(const uint64_t *a) {
    for (int i = 0; i < 4; i++) {
        if (a[i] != 0ULL) return 0;
    }
    return 1;
}

__device__ int bignum_is_odd(const uint64_t *a) {
    return a[0] & 1ULL;
}

__device__ void bignum_copy(uint64_t *dst, const uint64_t *src) {
    for (int i = 0; i < 4; i++) {
        dst[i] = src[i];
    }
}

__device__ void bignum_zero(uint64_t *a) {
    for (int i = 0; i < 4; i++) {
        a[i] = 0ULL;
    }
}

__device__ int bignum_is_one(const uint64_t *a) {
    if (a[0] != 1ULL) return 0;
    for (int i = 1; i < 4; i++) {
        if (a[i] != 0ULL) return 0;
    }
    return 1;
}

__device__ void bignum_set_ui(uint64_t *a, uint64_t val) {
    bignum_zero(a);
    a[0] = val;
}

__device__ uint64_t bignum_add_carry(uint64_t *result, const uint64_t *a, const uint64_t *b) {
    uint64_t carry = 0ULL;
    for (int i = 0; i < 4; i++) {
        uint64_t temp = a[i] + b[i] + carry;
        carry = (temp < a[i]) || (carry && temp == a[i]);
        result[i] = temp;
    }
    return carry;
}

__device__ uint64_t bignum_sub_borrow(uint64_t *result, const uint64_t *a, const uint64_t *b) {
    uint64_t borrow = 0ULL;
    for (int i = 0; i < 4; i++) {
        uint64_t temp = a[i] - b[i] - borrow;
        borrow = (a[i] < b[i] + borrow);
        result[i] = temp;
    }
    return borrow;
}

__device__ void bignum_shr1(uint64_t *result, const uint64_t *a) {
    uint64_t carry = 0ULL;
    for (int i = 3; i >= 0; i--) {
        uint64_t next = (a[i] & 1ULL) << 63;
        result[i] = (a[i] >> 1) | carry;
        carry = next;
    }
}

__device__ void bignum_mul_full(uint64_t *result_high, uint64_t *result_low,
                                const uint64_t *a, const uint64_t *b) {
    uint64_t temp_low[8] = {0};
    uint64_t temp_high[8] = {0};

    for (int i = 0; i < 4; i++) {
        uint64_t carry = 0ULL;
        for (int j = 0; j < 4; j++) {
            uint64_t a_hi = a[i] >> 32, a_lo = a[i] & 0xFFFFFFFFULL;
            uint64_t b_hi = b[j] >> 32, b_lo = b[j] & 0xFFFFFFFFULL;

            uint64_t p0 = a_lo * b_lo;
            uint64_t p1 = a_lo * b_hi;
            uint64_t p2 = a_hi * b_lo;
            uint64_t p3 = a_hi * b_hi;

            uint64_t mid = (p0 >> 32) + (p1 & 0xFFFFFFFFULL) + (p2 & 0xFFFFFFFFULL);
            uint64_t high = p3 + (p1 >> 32) + (p2 >> 32) + (mid >> 32);

            uint64_t low = (mid << 32) | (p0 & 0xFFFFFFFFULL);

            uint64_t sum_low = temp_low[i + j] + low + carry;
            carry = (sum_low < temp_low[i + j]) || (sum_low < low);
            temp_low[i + j] = sum_low;

            temp_high[i + j] += high + carry;
        }
        temp_high[i + 4] += carry;
    }

    for (int i = 0; i < 4; i++) {
        result_low[i] = temp_low[i];
        result_high[i] = temp_high[i];
    }
}

__device__ void montgomery_reduce_p(uint64_t *result,
                                    const uint64_t *input_high,
                                    const uint64_t *input_low) {
    uint64_t temp[8];

    for (int i = 0; i < 4; i++) {
        temp[i]     = input_low[i];
        temp[i + 4] = input_high[i];
    }

    for (int i = 0; i < 4; i++) {
        uint64_t ui = temp[i] * (uint64_t)MU_P;

        uint64_t carry = 0ULL;
        for (int j = 0; j < 4; j++) {
            unsigned __int128 prod =
                (unsigned __int128)ui * (unsigned __int128)P_CONST[j] +
                (unsigned __int128)temp[i + j] +
                (unsigned __int128)carry;

            temp[i + j] = (uint64_t)prod;
            carry       = (uint64_t)(prod >> 64);
        }

        for (int j = i + 4; j < 8; j++) {
            unsigned __int128 tmp = (unsigned __int128)temp[j] +
                                    (unsigned __int128)carry;

            temp[j] = (uint64_t)tmp;
            carry   = (uint64_t)(tmp >> 64);
        }
    }

    for (int i = 0; i < 4; i++) {
        result[i] = temp[i + 4];
    }

    if (bignum_cmp(result, (const uint64_t*)P_CONST) >= 0) {
        bignum_sub_borrow(result, result, (const uint64_t*)P_CONST);
    }
}

__device__ void to_montgomery_p(uint64_t *result, const uint64_t *a) {
    uint64_t high[4], low[4];
    bignum_mul_full(high, low, a, (uint64_t*)R2_MOD_P);
    montgomery_reduce_p(result, high, low);
}

__device__ void from_montgomery_p(uint64_t *result, const uint64_t *a) {
    uint64_t zero[4] = {0, 0, 0, 0};
    bignum_zero(zero);
    montgomery_reduce_p(result, zero, a);
}

__device__ void mod_add_p(uint64_t *result, const uint64_t *a, const uint64_t *b) {
    uint64_t temp[4];
    uint64_t carry = bignum_add_carry(temp, a, b);

    if (carry || bignum_cmp(temp, (uint64_t*)P_CONST) >= 0) {
        bignum_sub_borrow(result, temp, (uint64_t*)P_CONST);
    } else {
        bignum_copy(result, temp);
    }
}

__device__ void mod_sub_p(uint64_t *result, const uint64_t *a, const uint64_t *b) {
    uint64_t temp[4];
    uint64_t borrow = bignum_sub_borrow(temp, a, b);

    if (borrow) {
        bignum_add_carry(result, temp, (uint64_t*)P_CONST);
    } else {
        bignum_copy(result, temp);
    }
}

__device__ void mod_mul_mont_p(uint64_t *result, const uint64_t *a, const uint64_t *b) {
    uint64_t high[4], low[4];
    bignum_mul_full(high, low, a, b);
    montgomery_reduce_p(result, high, low);
}

/*

__device__ void mod_sqr_mont_p(uint64_t *result, const uint64_t *a) {
    mod_mul_mont_p(result, a, a);
}

static __device__ __forceinline__ bool is_zero_4(const uint64_t *x) {
    return (x[0] | x[1] | x[2] | x[3]) == 0ULL;
}

static __device__ __forceinline__ void copy_4(uint64_t *dst, const uint64_t *src) {
    dst[0] = src[0]; dst[1] = src[1]; dst[2] = src[2]; dst[3] = src[3];
}

static __device__ __forceinline__ void set_ui_4(uint64_t *dst, uint64_t g) {
    dst[0] = g; dst[1] = 0ULL; dst[2] = 0ULL; dst[3] = 0ULL;
}

static __device__ __forceinline__ void zero_4(uint64_t *dst) {
    dst[0] = dst[1] = dst[2] = dst[3] = 0ULL;
}

static __device__ __forceinline__ void shr1_4(uint64_t *x) {
    uint64_t carry = 0ULL;
    for (int k = 0; k < 4; ++k) {
        uint64_t next = (k < 3) ? ((x[k+1] & 1ULL) << 63) : 0ULL;
        uint64_t cur = x[k];
        x[k] = (cur >> 1) | carry;
        carry = next;
    }
}

static __device__ __forceinline__ void sub_and_shr1_4(uint64_t *res, const uint64_t *a, const uint64_t *b) {
    uint64_t tmp[4];
    uint64_t borrow = 0ULL;
    for (int i = 0; i < 4; ++i) {
        uint64_t bi = b[i] + borrow;
        tmp[i] = a[i] - bi;
        borrow = (a[i] < bi) ? 1ULL : 0ULL;
    }
    shr1_4(tmp);
    for (int i = 0; i < 4; ++i) res[i] = tmp[i];
}

static __device__ __forceinline__ void add_cond_4(uint64_t *dst, const uint64_t *src, uint64_t mask) {
    uint64_t carry = 0ULL;
    for (int t = 0; t < 4; ++t) {
        uint64_t s = src[t] & mask;
        uint64_t old = dst[t];
        uint64_t tmp = old + s;
        uint64_t carry1 = (tmp < old) ? 1ULL : 0ULL;
        uint64_t sum = tmp + carry;
        uint64_t carry2 = (sum < tmp) ? 1ULL : 0ULL;
        dst[t] = sum;
        carry = carry1 | carry2;
    }
}

static __device__ __forceinline__ void mul_4x4(uint64_t *res_low, uint64_t *res_high, const uint64_t *a, const uint64_t *b) {
    uint64_t tmp[8] = {0};
    for(int i=0;i<4;i++) {
        uint64_t carry = 0;
        for(int j=0;j<4;j++) {
            __uint128_t prod = (__uint128_t)a[i]*b[j] + tmp[i+j] + carry;
            tmp[i+j] = (uint64_t)prod;
            carry = (uint64_t)(prod >> 64);
        }
        tmp[i+4] = carry;
    }
    for(int k=0;k<4;k++) res_low[k] = tmp[k];
    for(int k=0;k<4;k++) res_high[k] = tmp[k+4];
}

static __device__ __forceinline__ void sub_4(uint64_t *res, const uint64_t *a, const uint64_t *b) {
    uint64_t borrow = 0ULL;
    for (int i = 0; i < 4; ++i) {
        uint64_t bi = b[i] + borrow;
        uint64_t ri = a[i] - bi;
        borrow = (a[i] < bi) ? 1ULL : 0ULL;
        res[i] = ri;
    }
}

static __device__ __forceinline__ void lshift1_4(uint64_t *dst, const uint64_t *src) {
    uint64_t carry = 0ULL;
    for (int k = 0; k < 4; ++k) {
        uint64_t cur = src[k];
        uint64_t next_carry = cur >> 63;
        dst[k] = (cur << 1) | carry;
        carry = next_carry;
    }
}

static __device__ __forceinline__ void transition_matrix_4(
    int32_t *delta, uint64_t *u, uint64_t *v, uint64_t t[16], int N)
{
    uint64_t m00[4], m01[4], m10[4], m11[4];
    set_ui_4(m00, 1ULL); zero_4(m01);
    zero_4(m10); set_ui_4(m11, 1ULL);

    uint64_t u_copy[4], v_copy[4];
    copy_4(u_copy, u); copy_4(v_copy, v);

    for (int i = 0; i < N; i++) {
        uint64_t v_odd = v_copy[0] & 1ULL;
        uint64_t delta_pos = (uint64_t)(*delta > 0);

        uint64_t cond0 = (uint64_t)(delta_pos & v_odd);
        uint64_t cond1 = (uint64_t)((!delta_pos) & v_odd);
        uint64_t cond2 = (uint64_t)(!v_odd);

        uint64_t mask0 = 0ULL - cond0;
        uint64_t mask1 = 0ULL - cond1;
        uint64_t mask2 = 0ULL - cond2;

        int32_t delta_new0 = 1 - *delta;
        int32_t delta_new1 = *delta + 1;
        int32_t delta_new2 = *delta + 1;
        *delta = (delta_new0 & (int32_t)mask0) |
                 (delta_new1 & (int32_t)mask1) |
                 (delta_new2 & (int32_t)mask2);

        uint64_t u_tmp[4], v_tmp[4];
        uint64_t sub0[4], sub1[4];

        copy_4(u_tmp, v_copy);
        sub_4(sub0, u_copy, v_copy); shr1_4(sub0); copy_4(v_tmp, sub0);
        sub_4(sub1, v_copy, u_copy); shr1_4(sub1);
        uint64_t v_half[4]; copy_4(v_half, v_copy); shr1_4(v_half);

        for (int w = 0; w < 4; w++) {
            u_copy[w] = (u_tmp[w] & mask0) | (u_copy[w] & ~mask0);
            v_copy[w] = (v_tmp[w] & mask0) |
                        (sub1[w]   & mask1) |
                        (v_half[w] & mask2);
        }

        uint64_t new_m00[4], new_m01[4], new_m10[4], new_m11[4];
        lshift1_4(new_m00, (cond0 ? m10 : m00));
        lshift1_4(new_m01, (cond0 ? m11 : m01));

        uint64_t tmp0[4], tmp1[4];
        sub_4(tmp0, m00, m10);
        sub_4(tmp1, m01, m11);

        uint64_t tmp2[4], tmp3[4];
        sub_4(tmp2, m10, m00);
        sub_4(tmp3, m11, m01);

        for (int w = 0; w < 4; w++) {
            uint64_t t0 = tmp0[w] & mask0;
            uint64_t t1 = tmp2[w] & mask1;
            uint64_t t2 = m10[w] & ~(mask0 | mask1);
            new_m10[w] = t0 | t1 | t2;

            uint64_t s0 = tmp1[w] & mask0;
            uint64_t s1 = tmp3[w] & mask1;
            uint64_t s2 = m11[w] & ~(mask0 | mask1);
            new_m11[w] = s0 | s1 | s2;
        }

        copy_4(m00, new_m00);
        copy_4(m01, new_m01);
        copy_4(m10, new_m10);
        copy_4(m11, new_m11);
    }

    copy_4(&t[0],  m00);
    copy_4(&t[4],  m01);
    copy_4(&t[8],  m10);
    copy_4(&t[12], m11);

    copy_4(u, u_copy);
    copy_4(v, v_copy);
}

static __device__ __forceinline__ void div2n_4(uint64_t *res, const uint64_t *x_low, const uint64_t *x_high, const uint64_t *p, const uint64_t *p_inv, int N) {

    uint64_t maskN = (N == 64) ? ~0ULL : ((1ULL << N) - 1ULL);

    uint64_t m0 = (x_low[0] * p_inv[0]) & maskN;

    uint64_t prod[5];
    __uint128_t carry_prod = 0;
    for (int i = 0; i < 4; ++i) {
        __uint128_t t = (__uint128_t)m0 * p[i] + carry_prod;
        prod[i] = (uint64_t)t;
        carry_prod = t >> 64;
    }
    prod[4] = (uint64_t)carry_prod;

    uint64_t full[8];
    uint64_t borrow = 0ULL;

    for (int i = 0; i < 4; ++i) {
        __uint128_t sum = (__uint128_t)prod[i] + (__uint128_t)borrow;
        uint64_t sum_lo = (uint64_t)sum;
        full[i] = x_low[i] - sum_lo;
        borrow = (x_low[i] < sum_lo) ? 1ULL : 0ULL;
    }

    __uint128_t sumh = (__uint128_t)prod[4] + (__uint128_t)borrow;
    uint64_t sumh_lo = (uint64_t)sumh;
    full[4] = x_high[0] - sumh_lo;
    borrow = (x_high[0] < sumh_lo) ? 1ULL : 0ULL;

    for (int i = 1; i < 4; ++i) {
        uint64_t sub = borrow;
        full[4 + i] = x_high[i] - sub;
        borrow = (x_high[i] < sub) ? 1ULL : 0ULL;
    }

    int wshift = N / 64;
    int bshift = N % 64;
    uint64_t shifted[8];

    if (bshift == 0) {
        for (int i = 0; i < 8; ++i) {
            int idx = i + wshift;
            shifted[i] = (idx < 8) ? full[idx] : 0ULL;
        }
    } else {
        for (int i = 0; i < 8; ++i) {
            int idx = i + wshift;
            uint64_t low = (idx < 8) ? full[idx] : 0ULL;
            uint64_t high = (idx + 1 < 8) ? full[idx + 1] : 0ULL;
            shifted[i] = (low >> bshift) | (high << (64 - bshift));
        }
    }

    for (int i = 0; i < 4; ++i) res[i] = shifted[i];
}

static __device__ __forceinline__ void update_x1x2_optimized_ver2_4(
    uint64_t *x1, uint64_t *x2,
    const uint64_t t[16],
    const uint64_t *p, const uint64_t *p_inv, int N)
{
    uint64_t x1n_low[4], x1n_high[4], x2n_low[4], x2n_high[4];
    uint64_t tmp_low[4], tmp_high[4];

    mul_4x4(x1n_low, x1n_high, x1, &t[0]);
    mul_4x4(tmp_low, tmp_high, x2, &t[4]);
    uint64_t carry = 0;
    for (int i = 0; i < 4; i++) {
        __uint128_t s = (__uint128_t)x1n_low[i] + tmp_low[i] + carry;
        x1n_low[i] = (uint64_t)s;
        carry = (uint64_t)(s >> 64);
    }

    mul_4x4(x2n_low, x2n_high, x1, &t[8]);
    mul_4x4(tmp_low, tmp_high, x2, &t[12]);
    carry = 0;
    for (int i = 0; i < 4; i++) {
        __uint128_t s = (__uint128_t)x2n_low[i] + tmp_low[i] + carry;
        x2n_low[i] = (uint64_t)s;
        carry = (uint64_t)(s >> 64);
    }

    div2n_4(x1, x1n_low, x1n_high, p, p_inv, N);
    div2n_4(x2, x2n_low, x2n_high, p, p_inv, N);

    uint64_t neg_mask_x1 = 0ULL - (x1[3] >> 63);
    add_cond_4(x1, p, neg_mask_x1);
    uint64_t neg_mask_x2 = 0ULL - (x2[3] >> 63);
    add_cond_4(x2, p, neg_mask_x2);
}

static __device__ __forceinline__ void normalize_4(
    uint64_t *res, uint64_t *v, int32_t sign, const uint64_t *p)
{
    uint64_t neg_mask = 0ULL - (v[3] >> 63);
    add_cond_4(v, p, neg_mask);

    uint64_t sign_mask = 0ULL - (uint64_t)(sign < 0);
    uint64_t carry = sign_mask & 1ULL;
    for (int i = 0; i < 4; i++) {
        uint64_t inv = (v[i] ^ sign_mask);
        __uint128_t s = (__uint128_t)inv + carry;
        v[i] = (uint64_t)s;
        carry = (uint64_t)(s >> 64);
    }

    neg_mask = 0ULL - (v[3] >> 63);
    add_cond_4(v, p, neg_mask);

    copy_4(res, v);
}
*/

__device__ void mod_sqr_mont_p(uint64_t *result, const uint64_t *a) {
    mod_mul_mont_p(result, a, a);
}

static __device__ __forceinline__ void copy_4(uint64_t *dst, const uint64_t *src) {
    dst[0] = src[0]; dst[1] = src[1]; dst[2] = src[2]; dst[3] = src[3];
}

static __device__ __forceinline__ void zero_4(uint64_t *dst) {
    dst[0] = dst[1] = dst[2] = dst[3] = 0ULL;
}

static __device__ __forceinline__ void set_ui_4(uint64_t *dst, uint64_t g) {
    dst[0] = g; dst[1] = dst[2] = dst[3] = 0ULL;
}

static __device__ __forceinline__ bool is_zero_4(const uint64_t *x) {
    return (x[0] | x[1] | x[2] | x[3]) == 0ULL;
}

static __device__ __forceinline__ void shr1_4(uint64_t *x) {
    uint64_t carry = 0ULL;
    for (int k = 0; k < 4; ++k) {
        uint64_t next = (k < 3) ? ((x[k+1] & 1ULL) << 63) : 0ULL;
        uint64_t cur = x[k];
        x[k] = (cur >> 1) | carry;
        carry = next;
    }
}

static __device__ __forceinline__ void mul_4x4(uint64_t *res_low, uint64_t *res_high, const uint64_t *a, const uint64_t *b) {
    uint64_t tmp[8] = {0};
    for(int i=0;i<4;i++) {
        uint64_t carry = 0;
        for(int j=0;j<4;j++) {
            __uint128_t prod = (__uint128_t)a[i]*b[j] + tmp[i+j] + carry;
            tmp[i+j] = (uint64_t)prod;
            carry = (uint64_t)(prod >> 64);
        }
        tmp[i+4] = carry;
    }
    for(int k=0;k<4;k++) res_low[k] = tmp[k];
    for(int k=0;k<4;k++) res_high[k] = tmp[k+4];
}

static __device__ __forceinline__ void sub_4(uint64_t *res, const uint64_t *a, const uint64_t *b) {
    uint64_t borrow = 0ULL;
    for (int i = 0; i < 4; ++i) {
        uint64_t bi = b[i] + borrow;
        uint64_t ri = a[i] - bi;
        borrow = (a[i] < bi) ? 1ULL : 0ULL;
        res[i] = ri;
    }
}

static __device__ __forceinline__ void lshift1_4(uint64_t *dst, const uint64_t *src) {
    uint64_t carry = 0ULL;
    for (int k = 0; k < 4; ++k) {
        uint64_t cur = src[k];
        uint64_t next_carry = cur >> 63;
        dst[k] = (cur << 1) | carry;
        carry = next_carry;
    }
}

static __device__ __forceinline__ void add_cond_4(uint64_t *dst, const uint64_t *src, uint64_t mask) {
    uint64_t carry = 0ULL;
    for (int i = 0; i < 4; ++i) {
        uint64_t s = src[i] & mask;
        uint64_t old = dst[i];
        __uint128_t sum = (__uint128_t)old + s + carry;
        dst[i] = (uint64_t)sum;
        carry = (uint64_t)(sum >> 64);
    }
}

static __device__ __forceinline__ void transition_matrix_4(
    int32_t *delta, uint64_t *u, uint64_t *v, uint64_t t[16], int N)
{
    uint64_t m00[4], m01[4], m10[4], m11[4];
    set_ui_4(m00, 1ULL); zero_4(m01);
    zero_4(m10); set_ui_4(m11, 1ULL);

    uint64_t u_copy[4], v_copy[4];
    copy_4(u_copy, u); copy_4(v_copy, v);

    for (int i = 0; i < N; i++) {
        uint64_t v_odd = v_copy[0] & 1ULL;
        uint64_t delta_pos = (uint64_t)(*delta > 0);

        uint64_t cond0 = delta_pos & v_odd;
        uint64_t cond1 = (~delta_pos) & v_odd;
        uint64_t cond2 = ~v_odd;

        uint64_t mask0 = 0ULL - cond0;
        uint64_t mask1 = 0ULL - cond1;
        uint64_t mask2 = 0ULL - cond2;

        int32_t delta_new0 = 1 - *delta;
        int32_t delta_new1 = *delta + 1;
        int32_t delta_new2 = *delta + 1;

        *delta = (delta_new0 & (int32_t)mask0) |
                 (delta_new1 & (int32_t)mask1) |
                 (delta_new2 & (int32_t)mask2);

        uint64_t u_tmp[4], v_tmp[4], sub0[4], sub1[4], v_half[4];
        copy_4(u_tmp, v_copy);
        sub_4(sub0, u_copy, v_copy); shr1_4(sub0); copy_4(v_tmp, sub0);
        sub_4(sub1, v_copy, u_copy); shr1_4(sub1);
        copy_4(v_half, v_copy); shr1_4(v_half);

        for (int w = 0; w < 4; w++) {
            u_copy[w] = (u_tmp[w] & mask0) | (u_copy[w] & ~mask0);
            v_copy[w] = (v_tmp[w] & mask0) |
                        (sub1[w]   & mask1) |
                        (v_half[w] & mask2);
        }

        uint64_t new_m00[4], new_m01[4], new_m10[4], new_m11[4];
        lshift1_4(new_m00, (cond0 ? m10 : m00));
        lshift1_4(new_m01, (cond0 ? m11 : m01));

        uint64_t tmp0[4], tmp1[4], tmp2[4], tmp3[4];
        sub_4(tmp0, m00, m10); sub_4(tmp1, m01, m11);
        sub_4(tmp2, m10, m00); sub_4(tmp3, m11, m01);

        for (int w = 0; w < 4; w++) {
            new_m10[w] = (tmp0[w] & mask0) | (tmp2[w] & mask1) | (m10[w] & ~(mask0 | mask1));
            new_m11[w] = (tmp1[w] & mask0) | (tmp3[w] & mask1) | (m11[w] & ~(mask0 | mask1));
        }

        copy_4(m00, new_m00); copy_4(m01, new_m01);
        copy_4(m10, new_m10); copy_4(m11, new_m11);
    }

    copy_4(&t[0],  m00); copy_4(&t[4],  m01);
    copy_4(&t[8],  m10); copy_4(&t[12], m11);
    copy_4(u, u_copy); copy_4(v, v_copy);
}

static __device__ __forceinline__ void div2n_4(uint64_t *res, const uint64_t *x_low, const uint64_t *x_high, const uint64_t *p, const uint64_t *p_inv, int N)
{
    uint64_t maskN = (N == 64) ? ~0ULL : ((1ULL << N) - 1ULL);
    uint64_t m0 = (x_low[0] * p_inv[0]) & maskN;

    uint64_t prod[5]; __uint128_t carry_prod = 0;
    for (int i = 0; i < 4; i++) {
        __uint128_t t = (__uint128_t)m0 * p[i] + carry_prod;
        prod[i] = (uint64_t)t;
        carry_prod = t >> 64;
    }
    prod[4] = (uint64_t)carry_prod;

    uint64_t full[8]; uint64_t borrow = 0;
    for (int i = 0; i < 4; i++) {
        __uint128_t sum = (__uint128_t)prod[i] + borrow;
        full[i] = x_low[i] - (uint64_t)sum;
        borrow = (x_low[i] < (uint64_t)sum) ? 1ULL : 0ULL;
    }
    __uint128_t sumh = (__uint128_t)prod[4] + borrow;
    full[4] = x_high[0] - (uint64_t)sumh;
    borrow = (x_high[0] < (uint64_t)sumh) ? 1ULL : 0ULL;
    for (int i = 1; i < 4; i++) { full[4+i] = x_high[i] - borrow; borrow = (x_high[i] < borrow) ? 1ULL : 0ULL; }

    int wshift = N/64, bshift = N%64;
    uint64_t shifted[8];
    if (bshift == 0) {
        for (int i = 0; i < 8; i++) shifted[i] = (i + wshift < 8) ? full[i + wshift] : 0ULL;
    } else {
        for (int i = 0; i < 8; i++) {
            uint64_t lo = (i + wshift < 8) ? full[i + wshift] : 0ULL;
            uint64_t hi = (i + wshift + 1 < 8) ? full[i + wshift + 1] : 0ULL;
            shifted[i] = (lo >> bshift) | (hi << (64-bshift));
        }
    }
    copy_4(res, shifted);
}

static __device__ __forceinline__ void update_x1x2_optimized_ver2_4(
    uint64_t *x1, uint64_t *x2,
    const uint64_t t[16],
    const uint64_t *p, const uint64_t *p_inv, int N)
{
    uint64_t x1n_low[4], x1n_high[4], x2n_low[4], x2n_high[4];
    uint64_t tmp_low[4], tmp_high[4];

    mul_4x4(x1n_low, x1n_high, x1, &t[0]);
    mul_4x4(tmp_low, tmp_high, x2, &t[4]);
    uint64_t carry = 0;
    for (int i = 0; i < 4; i++) {
        __uint128_t s = (__uint128_t)x1n_low[i] + tmp_low[i] + carry;
        x1n_low[i] = (uint64_t)s;
        carry = (uint64_t)(s >> 64);
    }

    mul_4x4(x2n_low, x2n_high, x1, &t[8]);
    mul_4x4(tmp_low, tmp_high, x2, &t[12]);
    carry = 0;
    for (int i = 0; i < 4; i++) {
        __uint128_t s = (__uint128_t)x2n_low[i] + tmp_low[i] + carry;
        x2n_low[i] = (uint64_t)s;
        carry = (uint64_t)(s >> 64);
    }

    div2n_4(x1, x1n_low, x1n_high, p, p_inv, N);
    div2n_4(x2, x2n_low, x2n_high, p, p_inv, N);

    uint64_t neg_mask_x1 = 0ULL - (x1[3] >> 63);
    add_cond_4(x1, p, neg_mask_x1);
    uint64_t neg_mask_x2 = 0ULL - (x2[3] >> 63);
    add_cond_4(x2, p, neg_mask_x2);
}

static __device__ __forceinline__ void normalize_4(uint64_t *res, uint64_t *v, int32_t sign, const uint64_t *p) {
    uint64_t neg_mask = 0ULL - (v[3] >> 63);
    add_cond_4(v, p, neg_mask);

    uint64_t sign_mask = 0ULL - (uint64_t)(sign < 0);
    uint64_t carry = sign_mask & 1ULL;
    for (int i = 0; i < 4; i++) {
        uint64_t inv = v[i] ^ sign_mask;
        __uint128_t s = (__uint128_t)inv + carry;
        v[i] = (uint64_t)s;
        carry = (uint64_t)(s >> 64);
    }

    neg_mask = 0ULL - (v[3] >> 63);
    add_cond_4(v, p, neg_mask);

    copy_4(res, v);
}

__device__ void mod_inverse_p(uint64_t *result, const uint64_t *a_normal) {

    const uint64_t p[4] = {
        0xFFFFFFFEFFFFFC2FULL,
        0xFFFFFFFFFFFFFFFFULL,
        0xFFFFFFFFFFFFFFFFULL,
        0xFFFFFFFFFFFFFFFFULL
    };

    const int N = 62;
    const uint64_t p_inv[4] = {0x27c7f6e22ddacacfULL, 0x0ULL, 0x0ULL, 0x0ULL}; // Compute x/2^N mod p, given p_inv = 1/p mod 2^N.

    if (is_zero_4(a_normal)) { zero_4(result); return; }

    int32_t delta = 1;
    const int d = 256;
    const int m = (49*d + 57)/17;

    /* --- From Paper (Daniel J. Bernstein and Bo-Yin Yang)
    from divsteps2 import divsteps2
    def iterations(d):
        return (49*d+80)//17 if d<46 else (49*d+57)//17
    */

    uint64_t f[4], g[4], x1[4], x2[4], t[16];
    copy_4(f, a_normal); copy_4(g, p);
    zero_4(x1); set_ui_4(x2, 1ULL);

    for (int i = 0; i < m; i++) {
        uint64_t g_odd = g[0] & 1ULL;
        uint64_t swap_mask = 0ULL - (uint64_t)(delta > 0 && g_odd);
        uint64_t inv_mask = ~swap_mask;

        uint64_t f_tmp[4], g_tmp[4], x1_tmp[4], x2_tmp[4];
        copy_4(f_tmp, f); copy_4(g_tmp, g);
        copy_4(x1_tmp, x1); copy_4(x2_tmp, x2);

        for (int k = 0; k < 4; k++) {
            f[k]  = (g_tmp[k] & swap_mask) | (f_tmp[k] & inv_mask);
            g[k]  = (f_tmp[k] & swap_mask) | (g_tmp[k] & inv_mask);
            x1[k] = (x2_tmp[k] & swap_mask) | (x1_tmp[k] & inv_mask);
            x2[k] = (x1_tmp[k] & swap_mask) | (x2_tmp[k] & inv_mask);
        }

        int32_t delta_new0 = 1 - delta;
        int32_t delta_new1 = delta + 1;
        delta = (delta_new0 & swap_mask) | (delta_new1 & inv_mask);

        uint64_t g_odd_mask = 0ULL - g_odd;
        add_cond_4(g, f, g_odd_mask);
        add_cond_4(x2, x1, g_odd_mask);
        shr1_4(g);

        transition_matrix_4(&delta, f, g, t, N);

        update_x1x2_optimized_ver2_4(x1, x2, t, p, p_inv, N);
    }

    normalize_4(result, x2, (delta > 0) ? 1 : -1, p);
}

__device__ void jacobian_init(ECPointJacobian *point) {
    bignum_zero(point->X);
    bignum_zero(point->Y);
    bignum_copy(point->Z, ONE_MONT);
    point->infinity = 0;
}

__device__ void jacobian_set_infinity(ECPointJacobian *point) {
    bignum_copy(point->X, ONE_MONT);
    bignum_copy(point->Y, ONE_MONT);
    bignum_zero(point->Z);
    point->infinity = 1;
}

__device__ int jacobian_is_infinity(const ECPointJacobian *point) {
    return point->infinity || bignum_is_zero(point->Z);
}

__device__ void affine_to_jacobian(ECPointJacobian *jac, const ECPoint *aff) {
    if (aff->infinity) {
        jacobian_set_infinity(jac);
        return;
    }
    
    bignum_copy(jac->X, aff->x);
    bignum_copy(jac->Y, aff->y);
    bignum_copy(jac->Z, ONE_MONT);
    jac->infinity = 0;
}

__device__ void jacobian_to_affine(ECPoint *aff, const ECPointJacobian *jac) {
    if (jacobian_is_infinity(jac)) {
        bignum_zero(aff->x);
        bignum_zero(aff->y);
        aff->infinity = 1;
        return;
    }

    uint64_t z_norm[4], z_inv[4], z_inv_sqr[4], z_inv_cube[4];

    from_montgomery_p(z_norm, jac->Z);

    mod_inverse_p(z_inv, z_norm);
    mod_mul_mont_p(z_inv_sqr, z_inv, z_inv);
    mod_mul_mont_p(z_inv_cube, z_inv_sqr, z_inv);
    mod_mul_mont_p(aff->x, jac->X, z_inv_sqr);
    mod_mul_mont_p(aff->y, jac->Y, z_inv_cube);

    from_montgomery_p(aff->x, aff->x);
    from_montgomery_p(aff->y, aff->y);

    aff->infinity = 0;
}

__device__ void jacobian_double(ECPointJacobian *result, const ECPointJacobian *point) {
    if (jacobian_is_infinity(point) || bignum_is_zero(point->Y)) {
        jacobian_set_infinity(result);
        return;
    }

    uint64_t A[4], B[4], C[4], D[4], E[4];
    uint64_t X2[4];

    mod_sqr_mont_p(A, point->Y);
    mod_mul_mont_p(B, point->X, A);
    mod_add_p(B, B, B);
    mod_add_p(B, B, B);

    mod_sqr_mont_p(C, A);
    mod_add_p(C, C, C);
    mod_add_p(C, C, C);
    mod_add_p(C, C, C);

    mod_sqr_mont_p(X2, point->X);
    mod_add_p(D, X2, X2);
    mod_add_p(D, D, X2);

    mod_sqr_mont_p(result->X, D);
    mod_sub_p(result->X, result->X, B);
    mod_sub_p(result->X, result->X, B);

    mod_sub_p(E, B, result->X);
    mod_mul_mont_p(result->Y, D, E);
    mod_sub_p(result->Y, result->Y, C);

    mod_mul_mont_p(result->Z, point->Y, point->Z);
    mod_add_p(result->Z, result->Z, result->Z);

    result->infinity = 0;
}

__device__ void jacobian_add(ECPointJacobian *result, const ECPointJacobian *P, const ECPointJacobian *Q) {

    int P_infinity = jacobian_is_infinity(P);
    int Q_infinity = jacobian_is_infinity(Q);

    if (P_infinity) {
        bignum_copy(result->X, Q->X);
        bignum_copy(result->Y, Q->Y);
        bignum_copy(result->Z, Q->Z);
        result->infinity = Q->infinity;
        return;
    }

    if (Q_infinity) {
        bignum_copy(result->X, P->X);
        bignum_copy(result->Y, P->Y);
        bignum_copy(result->Z, P->Z);
        result->infinity = P->infinity;
        return;
    }

    uint64_t U1[4], U2[4], S1[4], S2[4], H[4], I[4], J[4], r[4], V[4];
    uint64_t Z1Z1[4], Z2Z2[4], Z1Z2[4], temp1[4], temp2[4];

    mod_sqr_mont_p(Z1Z1, P->Z);
    mod_sqr_mont_p(Z2Z2, Q->Z);
    mod_mul_mont_p(U1, P->X, Z2Z2);
    mod_mul_mont_p(U2, Q->X, Z1Z1);
    mod_mul_mont_p(temp1, Q->Z, Z2Z2);
    mod_mul_mont_p(S1, P->Y, temp1);
    mod_mul_mont_p(temp2, P->Z, Z1Z1);
    mod_mul_mont_p(S2, Q->Y, temp2);
    mod_sub_p(H, U2, U1);
    mod_sub_p(r, S2, S1);

    int is_H_zero = (bignum_cmp(H, ZERO) == 0);
    int is_r_zero = (bignum_cmp(r, ZERO) == 0);

    if (is_H_zero) {
        if (is_r_zero) {
            jacobian_double(result, P);
        } else {
            jacobian_set_infinity(result);
        }
        return;
    }

    mod_add_p(I, H, H);
    mod_sqr_mont_p(I, I);
    mod_mul_mont_p(J, H, I);
    mod_mul_mont_p(V, U1, I);
    mod_add_p(r, r, r);
    mod_sqr_mont_p(result->X, r);
    mod_sub_p(result->X, result->X, J);
    mod_sub_p(result->X, result->X, V);
    mod_sub_p(result->X, result->X, V);
    mod_sub_p(temp1, V, result->X);
    mod_mul_mont_p(result->Y, r, temp1);
    mod_mul_mont_p(temp2, S1, J);
    mod_add_p(temp2, temp2, temp2);
    mod_sub_p(result->Y, result->Y, temp2);
    mod_add_p(Z1Z2, P->Z, Q->Z);
    mod_sqr_mont_p(Z1Z2, Z1Z2);
    mod_sub_p(Z1Z2, Z1Z2, Z1Z1);
    mod_sub_p(Z1Z2, Z1Z2, Z2Z2);
    mod_mul_mont_p(result->Z, Z1Z2, H);

    result->infinity = 0;
}

__device__ void scalar_reduce_n(uint64_t *r, const uint64_t *k) {
    uint64_t t[4];
    uint64_t borrow = bignum_sub_borrow(t, k, (uint64_t*)N_CONST);

    if (borrow == 0) {
        bignum_copy(r, t);
    } else {
        bignum_copy(r, k);
    }
}

__device__ void jacobian_scalar_mult(ECPointJacobian *result, const uint64_t *scalar, const ECPointJacobian *point) {
    if (bignum_is_zero(scalar) || jacobian_is_infinity(point)) {
        jacobian_set_infinity(result);
        return;
    }

    ECPointJacobian R0, R1;
    jacobian_set_infinity(&R0);
    R1 = *point;

    uint64_t k[4];
    bignum_copy(k, scalar);

    scalar_reduce_n(k, k);

    int msb = 255;
    while (msb >= 0) {
        int word = 3 - (msb / 64);
        int bit  = 63 - (msb % 64);
        if ((k[word] >> bit) & 1ULL) break;
        msb--;
    }

    for (int i = msb; i >= 0; i--) {
        int word = 3 - (i / 64);
        int bit  = 63 - (i % 64);
        int kbit = (k[word] >> bit) & 1ULL;

        if (kbit == 0) {
            ECPointJacobian temp;
            jacobian_add(&temp, &R1, &R0);
            R1 = temp;
            jacobian_double(&R0, &R0);
        } else {
            ECPointJacobian temp;
            jacobian_add(&temp, &R0, &R1);
            R0 = temp;
            jacobian_double(&R1, &R1);
        }
    }

    *result = R0;
}

__device__ void point_from_montgomery(ECPoint *result, const ECPoint *point_mont) {
    if (point_mont->infinity) {
        result->infinity = 1;
        bignum_zero(result->x);
        bignum_zero(result->y);
        return;
    }
    
    from_montgomery_p(result->x, point_mont->x);
    from_montgomery_p(result->y, point_mont->y);
    result->infinity = 0;
}

__device__ void kernel_point_init(ECPoint *point) {
    bignum_zero(point->x);
    bignum_zero(point->y);
    point->infinity = 0;
}

__device__ void kernel_point_add(ECPoint *R, const ECPoint *P, const ECPoint *Q) {
    ECPointJacobian P_jac, Q_jac, R_jac;
    
    affine_to_jacobian(&P_jac, P);
    affine_to_jacobian(&Q_jac, Q);    
    jacobian_add(&R_jac, &P_jac, &Q_jac);   
    jacobian_to_affine(R, &R_jac);
}

__device__ void kernel_point_double(ECPoint *R, const ECPoint *P) {
    ECPointJacobian P_jac, R_jac;
    
    affine_to_jacobian(&P_jac, P);
    jacobian_double(&R_jac, &P_jac);
    jacobian_to_affine(R, &R_jac);
}

__device__ void kernel_scalar_mult(ECPoint *R, const uint64_t *k, const ECPoint *P) {
    ECPointJacobian P_jac, R_jac;

    affine_to_jacobian(&P_jac, P);
    jacobian_scalar_mult(&R_jac, k, &P_jac);
    jacobian_to_affine(R, &R_jac);
}

__device__ int kernel_point_is_valid(const ECPoint *point) {
    if (point->infinity) return 1;

    uint64_t lhs[4], rhs[4];

    mod_sqr_mont_p(lhs, point->y);
    mod_sqr_mont_p(rhs, point->x);
    mod_mul_mont_p(rhs, rhs, point->x);
    mod_add_p(rhs, rhs, SEVEN_MONT);

    return (bignum_cmp(lhs, rhs) == 0);
}

__device__ void kernel_get_compressed_public_key(unsigned char *out, const ECPoint *public_key) {
    unsigned char prefix = (public_key->y[0] & 1ULL) ? 0x03 : 0x02;
    out[0] = prefix;

    for (int i = 0; i < 4; i++) {
        uint64_t word = public_key->x[3 - i];
        out[1 + i*8 + 0] = (word >> 56) & 0xFF;
        out[1 + i*8 + 1] = (word >> 48) & 0xFF;
        out[1 + i*8 + 2] = (word >> 40) & 0xFF;
        out[1 + i*8 + 3] = (word >> 32) & 0xFF;
        out[1 + i*8 + 4] = (word >> 24) & 0xFF;
        out[1 + i*8 + 5] = (word >> 16) & 0xFF;
        out[1 + i*8 + 6] = (word >> 8)  & 0xFF;
        out[1 + i*8 + 7] = word & 0xFF;
    }
}

__global__ void generate_public_key(unsigned char *out, const uint64_t *PRIV_KEY) {
    ECPoint pub;
    ECPoint G;
    ECPointJacobian G_jac, pub_jac;

    to_montgomery_p(G.x, GX_CONST);
    to_montgomery_p(G.y, GY_CONST);
    G.infinity = 0;

    affine_to_jacobian(&G_jac, &G);
    jacobian_scalar_mult(&pub_jac, PRIV_KEY, &G_jac);
    jacobian_to_affine(&pub, &pub_jac);

    kernel_get_compressed_public_key(out, &pub);
}

__global__ void point_init(ECPoint *point) {
    kernel_point_init(point);
}

__global__ void point_add(ECPoint *R, const ECPoint *P, const ECPoint *Q) {
    kernel_point_add(R, P, Q);
}

__global__ void point_double(ECPoint *R, const ECPoint *P) {
    kernel_point_double(R, P);
}

__global__ void scalar_mult(ECPoint *R, const uint64_t *k, const ECPoint *P) {
    kernel_scalar_mult(R, k, P);
}

__global__ void point_is_valid(int *result, const ECPoint *point) {
    *result = kernel_point_is_valid(point);
}

__global__ void get_compressed_public_key(unsigned char *out, const ECPoint *pub) {
    kernel_get_compressed_public_key(out, pub);
}

__global__ void test_inverse_kernel(uint64_t *a, uint64_t *result) {
    mod_inverse_p(result, a);
}

int main() {

    //LSB 130 Bits:
    uint64_t h_priv[4] = {
    0x28b88cf897c603c9ULL,
    0x3e7665705359f04fULL,
    0x0000000000000003ULL,
    0x0000000000000000ULL
    };

    /* h_priv ≡ 1 (mod p): */
    //Hex: 7FDB62ED2D6FA0874ABD664C95B7CEF2ED79CC82D13FF3AC8E9766AA21BEBEAE
    //Dec: 57831354042695616917422878622316954017183908093256327737334808907053491207854

    uint64_t h_result[4];
    uint64_t *d_priv = nullptr;
    uint64_t *d_result = nullptr;

    hipMalloc((void**)&d_priv, 4 * sizeof(uint64_t));
    hipMalloc((void**)&d_result, 4 * sizeof(uint64_t));
    hipMemcpy(d_priv, h_priv, 4 * sizeof(uint64_t), hipMemcpyHostToDevice);

    test_inverse_kernel<<<1,1>>>(d_priv, d_result);
    hipDeviceSynchronize();
    hipMemcpy(h_result, d_result, 4 * sizeof(uint64_t), hipMemcpyDeviceToHost);

    //Print in MSB
    printf("%016llx%016llx%016llx%016llx\n",
           h_result[3], h_result[2], h_result[1], h_result[0]);

    hipFree(d_priv);
    hipFree(d_result);

    hipDeviceReset();
    return 0;
}






